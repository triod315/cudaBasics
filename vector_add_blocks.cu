
#include <hip/hip_runtime.h>
#include <iostream>

__global__ void add(int*a, int*b, int*c) 
{
    c[blockIdx.x] = a[blockIdx.x] + b[blockIdx.x];
}

void random_ints(int* a, int N)
{
   int i;
   for (i = 0; i < N; ++i)
    {
        a[i] = rand()%1000;
    }
}

#define N 1000000000
int main(void){
    int *a,*b,*c;  //host variables
    int *d_a, *d_b, *d_c; //device variables
    int size=N*sizeof(int);

    //allocate space on device for a,b and c
    hipMalloc((void**)&d_a, size);
    hipMalloc((void**)&d_b, size);
    hipMalloc((void**)&d_c, size);

    a = (int *)malloc(size);
    random_ints(a, N);

    b = (int *)malloc(size);
    random_ints(b, N);

    c = (int *)malloc(size);

    hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);

    //launch kernel for N blocks
    add<<<N,1>>>(d_a,d_b,d_c);
    
    hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);

    hipDeviceSynchronize();
    hipError_t error=hipGetLastError();

    if(error!=hipSuccess){
        printf("Error: %s\n",hipGetErrorString(error));
    }

    /*for (int i=0;i<N;i++)
    {
        std::cout<<a[i]<<"+"<<b[i]<<"="<<c[i]<<std::endl;
    }
    */
    std::cout<<"done\n";
    free(a);
    free(b);
    free(c);
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    return 0;
}
