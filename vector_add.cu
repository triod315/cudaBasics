
#include <hip/hip_runtime.h>
#include <iostream>

__global__ void add(int*a, int*b, int*c) {*c = *a + *b;}

int main(void){
    int a,b,c;  //host variables
    int *d_a, *d_b, *d_c; //device variables
    int size=sizeof(int);

    //allocate space on device for a,b and c
    hipMalloc((void**)&d_a, size);
    hipMalloc((void**)&d_b, size);
    hipMalloc((void**)&d_c, size);

    a=1;
    b=1;

    hipMemcpy(d_a, &a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, &b, size, hipMemcpyHostToDevice);

    add<<<1,1>>>(d_a,d_b,d_c);
    
    hipMemcpy(&c, d_c, size, hipMemcpyDeviceToHost);

    hipDeviceSynchronize();
    hipError_t error=hipGetLastError();

    if(error!=hipSuccess){
        printf("Error: %s\n",hipGetErrorString(error));
    }

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    printf("%d",c);
    return 0;
}
