#include<fstream>
#include<iostream>
#include<vector>
#include<ctime>
#include<hip/hip_runtime.h>

using namespace std;

int N,M;

#define THREADS_PER_BLOCK 512

vector<int> readVector(ifstream &fin)
{
    
    //fin.open();

    int n;
    int c;
    fin>>n;

    vector<int> result;
    for (int i=0;i<n;i++){
        fin>>c;
        result.push_back(c);        
    }
    N=n;
    return result;
}

__global__ void add(int*a, int*b, int*c,int n) {

    int index=threadIdx.x+blockIdx.x*blockDim.x;
    if (index<n)
        c[index] = a[index] * b[index];
}

void doIt(int* sample,ofstream &fout){
    clock_t begin=clock();

    int *a,*b,*c;  //host variables
    int *d_a, *d_b, *d_c; //device variables
    int size=N*sizeof(int);

    hipMalloc((void**)&d_a, size);
    hipMalloc((void**)&d_b, size);
    hipMalloc((void**)&d_c, size);

    //a = (int *)malloc(size);
    a=sample;

    //b = (int *)malloc(size);
    b=sample;

    c = (int *)malloc(size);

    hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);

    //launch kernel for N blocks
    add<<<(N+M-1)/M,M>>>(d_a,d_b,d_c,M);
    
    hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);

    hipDeviceSynchronize();
    hipError_t error=hipGetLastError();

    if(error!=hipSuccess){
        printf("Error: %s\n",hipGetErrorString(error));
    }

    /*for (int i=0;i<N;i++)
    {
        std::cout<<a[i]<<"+"<<b[i]<<"="<<c[i]<<std::endl;
    }
    */
    
    //free(a);
    //free(b);
    free(c);
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    clock_t end=clock();
    double elapsed_secs = double(end - begin) / CLOCKS_PER_SEC;
    cout<<". Elapsed time: "<<elapsed_secs<<endl;
    fout<<"{"<<N<<", "<<elapsed_secs<<"},";
}




int main(int argc, char ** argv)
{
    cout<<"file name: "<<argv[1]<<endl;
    cout<<"Sample count: "<<argv[2]<<endl;
    cout<<"Threads per block"<<THREADS_PER_BLOCK<<endl;
    string fileName=argv[1];
    int sample_count=stoi(argv[2]);

    M=1024;

    //cout<<"Sample count: "<<sample_count<<endl;
    vector<int> sample;

    ifstream fin(fileName);
    ofstream fout("result.txt");


    for (int i=0;i<sample_count;i++){
        cout<<"Sample №"<<i;
        sample=readVector(fin);
        doIt(&sample[0],fout);
        
    }
    fout.close();
    return 0;
}