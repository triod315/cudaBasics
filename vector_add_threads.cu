
#include <hip/hip_runtime.h>
#include <iostream>

__global__ void add(int*a, int*b, int*c) 
{
    c[threadIdx.x] = a[threadIdx.x] + b[threadIdx.x];
}

void random_ints(int* a, int N)
{
   int i;
   for (i = 0; i < N; ++i)
    {
        a[i] = rand()%1000;
    }
}

#define N 1000
int main(void){
    int *a,*b,*c;  //host variables
    int *d_a, *d_b, *d_c; //device variables
    int size=N*sizeof(int);

    //allocate space on device for a,b and c
    hipMalloc((void**)&d_a, size);
    hipMalloc((void**)&d_b, size);
    hipMalloc((void**)&d_c, size);

    a = (int *)malloc(size);
    random_ints(a, N);

    b = (int *)malloc(size);
    random_ints(b, N);

    c = (int *)malloc(size);

    hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);

    //launch kernel for N threads
    add<<<1,N>>>(d_a,d_b,d_c);
    
    hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);

    hipDeviceSynchronize();
    hipError_t error=hipGetLastError();

    if(error!=hipSuccess){
        printf("Error: %s\n",hipGetErrorString(error));
    }

    /*for (int i=0;i<N;i++)
    {
        std::cout<<a[i]<<"+"<<b[i]<<"="<<c[i]<<std::endl;
    }
    */
    std::cout<<"done\n";
    free(a);
    free(b);
    free(c);
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    return 0;
}
