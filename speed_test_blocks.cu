
#include <hip/hip_runtime.h>
#include<fstream>
#include<iostream>
#include<vector>
#include<ctime>

using namespace std;

int N;

vector<int> readVector(ifstream &fin)
{
    
    //fin.open();

    int n;
    int c;
    fin>>n;

    vector<int> result;
    for (int i=0;i<n;i++){
        fin>>c;
        result.push_back(c);        
    }
    N=n;
    return result;
}

__global__ void add(int*a, int*b, int*c) {
    c[blockIdx.x] = a[blockIdx.x] + b[blockIdx.x];
}

void doIt(int* sample,ofstream &fout){
    clock_t begin=clock();

    int *a,*b,*c;  //host variables
    int *d_a, *d_b, *d_c; //device variables
    int size=N*sizeof(int);

    hipMalloc((void**)&d_a, size);
    hipMalloc((void**)&d_b, size);
    hipMalloc((void**)&d_c, size);

    //a = (int *)malloc(size);
    a=sample;

    //b = (int *)malloc(size);
    b=sample;

    c = (int *)malloc(size);

    hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);

    //launch kernel for N blocks
    add<<<N,1>>>(d_a,d_b,d_c);
    
    hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);

    hipDeviceSynchronize();
    hipError_t error=hipGetLastError();

    if(error!=hipSuccess){
        printf("Error: %s\n",hipGetErrorString(error));
    }

    /*for (int i=0;i<N;i++)
    {
        std::cout<<a[i]<<"+"<<b[i]<<"="<<c[i]<<std::endl;
    }
    */
    cout<<endl<<N<<"|"<<a[N-1]<<"+"<<b[N-1]<<"="<<c[N-1]<<std::endl;
    std::cout<<"done\n";
    //free(a);
    //free(b);
    free(c);
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    clock_t end=clock();
    double elapsed_secs = double(end - begin) / CLOCKS_PER_SEC;
    cout<<". Elapsed time: "<<elapsed_secs<<endl;
    fout<<"{"<<N<<", "<<elapsed_secs<<"},";
}



int main(int argc, char ** argv)
{
    cout<<"file name: "<<argv[1]<<endl;
    cout<<"Sample count: "<<argv[2]<<endl;
    string fileName=argv[1];
    int sample_count=stoi(argv[2]);
    //cout<<"Sample count: "<<sample_count<<endl;
    vector<int> sample;

    ifstream fin(fileName);
    ofstream fout("result.txt");


    for (int i=0;i<sample_count;i++){
        cout<<"Sample №"<<i;
        sample=readVector(fin);
        doIt(&sample[0],fout);
        
    }
    fout.close();
    return 0;
}