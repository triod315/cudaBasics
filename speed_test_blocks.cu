
#include <hip/hip_runtime.h>
#include<fstream>
#include<iostream>
#include<vector>
#include<ctime>
#include<cmath>

using namespace std;

int N;

vector<float> readVector(ifstream &fin)
{
    
    //fin.open();

    int n;
    int c;
    fin>>n;

    vector<float> result;
    for (int i=0;i<n;i++){
        fin>>c;
        result.push_back(c);        
    }
    N=n;
    return result;
}

__global__ void add(float*a, float*b, float*c) {
    c[blockIdx.x] = sinf(cosf(sinf(a[blockIdx.x]))) + sinf(cosf(sinf(b[blockIdx.x])));
}

void doIt(float* sample,ofstream &fout){
    clock_t begin=clock();

    float *a,*b,*c;  //host variables
    float *d_a, *d_b, *d_c; //device variables
    int size=N*sizeof(float);

    hipMalloc((void**)&d_a, size);
    hipMalloc((void**)&d_b, size);
    hipMalloc((void**)&d_c, size);

    //a = (int *)malloc(size);
    a=sample;

    //b = (int *)malloc(size);
    b=sample;

    c = (float *)malloc(size);

    hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);

    //launch kernel for N blocks
    add<<<N,1>>>(d_a,d_b,d_c);
    
    hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);

    hipDeviceSynchronize();
    hipError_t error=hipGetLastError();

    if(error!=hipSuccess){
        printf("Error: %s\n",hipGetErrorString(error));
    }

    /*for (int i=0;i<N;i++)
    {
        std::cout<<a[i]<<"+"<<b[i]<<"="<<c[i]<<std::endl;
    }
    */
   
    //free(a);
    //free(b);
    free(c);
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    clock_t end=clock();
    double elapsed_secs = double(end - begin) / CLOCKS_PER_SEC;
    cout<<". Elapsed time: "<<elapsed_secs<<endl;
    fout<<"{"<<N<<", "<<elapsed_secs<<"},";
}



int main(int argc, char ** argv)
{
    cout<<"file name: "<<argv[1]<<endl;
    cout<<"Sample count: "<<argv[2]<<endl;
    string fileName=argv[1];
    int sample_count=stoi(argv[2]);
    //cout<<"Sample count: "<<sample_count<<endl;
    vector<float> sample;

    ifstream fin(fileName);
    ofstream fout("result.txt");


    for (int i=0;i<sample_count;i++){
        cout<<"Sample №"<<i;
        sample=readVector(fin);
        doIt(&sample[0],fout);
        
    }
    fout.close();
    return 0;
}